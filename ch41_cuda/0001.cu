
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void open_gpu() {
printf("Running program from my NVIDIA GeForce RTX 2060 GPU!\n");
}

int main() {
printf("Running program from my INTEL Core i7-10750H CPU!\n");

open_gpu<<<1,1>>>();
hipDeviceSynchronize();
    return 0;
}
